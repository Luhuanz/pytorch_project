#include "hip/hip_runtime.h"

#include "onnx-tensorrt/onnxplugin.hpp"

using namespace ONNXPlugin;

static __device__ float sigmoid(float x){
    return 1 / (1 + expf(-x));
}

static __global__ void MYSELU_kernel_fp32(const float* x, float* output, int edge) {

    int position = threadIdx.x + blockDim.x * blockIdx.x;
	if(position >= edge) return;

    output[position] = x[position] * sigmoid(x[position]);
}

class MYSELU : public TRTPlugin {
public:
	SetupPlugin(MYSELU);

	virtual void config_finish() override{
		printf("\033[33minit MYSELU config: %s\033[0m\n", config_->info_.c_str());
		printf("weights count is %d\n", config_->weights_.size());
	}

	int enqueue(const std::vector<GTensor>& inputs, std::vector<GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) override{
		
		int n = inputs[0].count();
		const int nthreads = 512;
		int block_size = n < nthreads ? n : nthreads;
		int grid_size = (n + block_size - 1) / block_size;

		MYSELU_kernel_fp32 <<<grid_size, block_size, 0, stream>>> (inputs[0].ptr<float>(), outputs[0].ptr<float>(), n);
		return 0;
	}
};

RegisterPlugin(MYSELU);